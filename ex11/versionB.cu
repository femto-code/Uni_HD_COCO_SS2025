#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 256

__device__ int data[N];
__device__ volatile int flag = 0;
__device__ int result = 0;

__device__ unsigned long long consumer_time = 0;

__global__ void versionB1() {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < N; ++i)
            data[i] = i * 10;

        __threadfence();  // Ensure data[] is visible before flag
        flag = 1;
    }

    if (blockIdx.x == 1 && threadIdx.x == 0) {
        while (flag == 0);

        // Start timing
        unsigned long long start = clock64();

        int sum = 0;
        for (int i = 0; i < N; ++i)
            sum += data[i];

        // End timing
        unsigned long long end = clock64();
        consumer_time = end - start;

        result = sum;
    }
}

__global__ void versionB2() {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < N; ++i)
            data[i] = i * 10;

        flag = 1;
    }

    if (blockIdx.x == 1 && threadIdx.x == 0) {
        while (flag == 0);

        __threadfence();

        // Start timing
        unsigned long long start = clock64();

        int sum = 0;
        for (int i = 0; i < N; ++i)
            sum += data[i];

        // End timing
        unsigned long long end = clock64();
        consumer_time = end - start;

        result = sum;
    }
}

__global__ void versionB3() {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < N; ++i)
            data[i] = i * 10;

        __threadfence();
        flag = 1;
    }

    if (blockIdx.x == 1 && threadIdx.x == 0) {
        while (flag == 0);

        __threadfence();

        // Start timing
        unsigned long long start = clock64();

        int sum = 0;
        for (int i = 0; i < N; ++i)
            sum += data[i];

        // End timing
        unsigned long long end = clock64();
        consumer_time = end - start;

        result = sum;
    }
}

void run_version(const char* label, void (*kernel)()) {
    int host_result = 0;
    int zero = 0;
    int expected = (N - 1) * N / 2 * 10;

    // Reset
    hipMemcpyToSymbol(HIP_SYMBOL(flag), &zero, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(result), &zero, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(consumer_time), &zero, sizeof(unsigned long long));

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    kernel<<<2, 1>>>();
    hipEventRecord(stop);
    hipDeviceSynchronize();

    hipMemcpyFromSymbol(&host_result, HIP_SYMBOL(result), sizeof(int));
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    unsigned long long host_consumer_time = 0;
    hipMemcpyFromSymbol(&host_consumer_time, HIP_SYMBOL(consumer_time), sizeof(unsigned long long));

    // Convert cycles to ms (approximate, using clock rate)
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    float consumer_ms = (float)host_consumer_time / prop.clockRate * 1000.0f;

    printf("%s => result: %d [%s], total time: %.3f ms, consumer loop: %.6f ms\n",
       label, host_result,
       (host_result == expected ? "OK" : "FAIL"), ms, consumer_ms);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    printf("=== Version B: Memory Fence Variants ===\n");

    run_version("Version B1: fence in producer", versionB1);
    run_version("Version B2: fence in consumer", versionB2);
    run_version("Version B3: fence in both", versionB3);

    return 0;
}
