#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 32

__device__ int result = 0;
__device__ unsigned long long consumer_time = 0;

__global__ void versionBonus() {
    int tid = threadIdx.x;
    int value = tid * 10;

    // Producer: thread 0
    if (tid == 0) {
        value = 12345; // Example message
    }

    // Broadcast value from thread 0 to all threads in the warp
    int msg = __shfl_sync(0xFFFFFFFF, value, 0);

    // Consumer: all threads receive the message
    unsigned long long start = clock64();
    int sum = msg; // For demonstration, sum is just the message
    unsigned long long end = clock64();

    if (tid == 0) {
        result = sum;
        consumer_time = end - start;
    }
}

int main() {
    int host_result = 0;
    int zero = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(result), &zero, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(consumer_time), &zero, sizeof(unsigned long long));

    versionBonus<<<1, N>>>();
    hipDeviceSynchronize();

    hipMemcpyFromSymbol(&host_result, HIP_SYMBOL(result), sizeof(int));
    unsigned long long host_consumer_time = 0;
    hipMemcpyFromSymbol(&host_consumer_time, HIP_SYMBOL(consumer_time), sizeof(unsigned long long));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    float consumer_ms = (float)host_consumer_time / prop.clockRate * 1000.0f;

    printf("Bonus (warp shuffle): result: %d, consumer loop: %.6f ms\n", host_result, consumer_ms);

    return 0;
}